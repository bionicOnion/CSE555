#include "hip/hip_runtime.h"
/*
 * main.cu
 * Author: Robert Miller
 * Last Edited: 2/2/16
 *
 * The main entry point of the application which will align and composite images representing the red, green, and blue
 *   color channels of a single image by employing image pyramids.
 *
 * Images are loaded and displayed using the utilities provided by OpenCV, but most of the processing work has been
 *   handed off to the GPU through the use of CUDA C. This has the marked disadvantage that this code can only run on
 *   systems with CUDA-compliant hardware (namely GPUs sold by Nvidia), but the framework has been used here as a
 *   learning tool. Future projects may be implemented using OpenCL to help ensure greater compatibility.
 */


#include <chrono>
#include <iostream>
#include <stdint.h>
#include <string>

// CUDA
#include <hip/hip_runtime.h>

// OpenCV
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>

#include "accelerator.cuh"
#include "constants.cuh"


// A macro for clean error handling without unduly cluttering the code
#define CUDA_CALL(CALL, ERR_CODE, LINE)																					\
{																														\
	hipError_t retCode = (CALL);																						\
	if (retCode != hipSuccess)																							\
	{																													\
		std::cout << "A CUDA call has failed on line " << (LINE) <<  ": " << hipGetErrorString(retCode) << std::endl;	\
		return (ERR_CODE);																								\
	}																													\
}


struct PyramidLevel
{
	unsigned int offset;
	short2 dims;
};


int DEBUG_displayImage(uint8_t* devPtr, short2 imgDims, char* title)
{
	auto buf = malloc(imgDims.x * imgDims.y);
	CUDA_CALL(hipMemcpy(buf, devPtr, imgDims.x * imgDims.y, hipMemcpyDeviceToHost), HOST_CPY_FAIL, __LINE__);
	cv::Mat img(imgDims.y, imgDims.x, CV_8UC1, buf);
	auto smallSize = imgDims.x > imgDims.y
		? cv::Size(MAX_SMALL_IMG_DIM, (short) (imgDims.y * (((float) MAX_SMALL_IMG_DIM) / imgDims.x)))
		: cv::Size((short) (imgDims.x * (((float) MAX_SMALL_IMG_DIM) / imgDims.y)), MAX_SMALL_IMG_DIM);
	if (imgDims.x < MAX_SMALL_IMG_DIM && imgDims.y < MAX_SMALL_IMG_DIM)
		smallSize = cv::Size(imgDims.x, imgDims.y);
	cv::resize(img, img, smallSize);
	cv::imshow(title, img);
	cv::waitKey();
	return SUCCESS;
}


int main(int argc, char** argv)
{
	// If too few arguments were provided, print a usage message and exit
	if (argc < EXPECTED_NUM_ARGS)
	{
		std::cout << "Too few arguments provided." << std::endl << "\tUsage: " << argv[PNAME_ARG_INDEX] <<
			" [image] [align mode]" << std::endl;
		return INCORRECT_USAGE;
	}

	// Process the provided arguments
	std::string imageName = argv[IMG_ARG_INDEX];
	std::string alignModeArg = argv[ALIGN_MODE_INDEX];
	bool multiLayerAlignMode;
	short2 alignmentWindow;
	if (alignModeArg == MULTI_LAYER_SPECIFIER)
	{
		multiLayerAlignMode = true;
	}
	else if (alignModeArg == SINGLE_LAYER_SPECIFIER)
	{
		if (argc < EXPECTED_NUM_ARGS_WINDOW)
		{
			std::cout << "To use the single-layer alignment mode, an alignment window must be specified." << std::endl;
			return INCORRECT_USAGE;
		}
		multiLayerAlignMode = false;

		std::string alignWindowX = argv[X_WINDOW_RANGE];
		std::string alignWindowY = argv[Y_WINDOW_RANGE];

		alignmentWindow = make_short2(std::stoi(alignWindowX, nullptr), std::stoi(alignWindowY, nullptr));
	}
	else
	{
		std::cout << "Unrecognized alignment mode " << alignModeArg << std::endl;
		return INCORRECT_USAGE;
	}

	// Record the time before starting computation
	auto startTime = std::chrono::high_resolution_clock::now();

	// Load the specified source image and calculate the dimensions for the resulting composite image
	auto sourceImage = cv::imread(imageName, CV_LOAD_IMAGE_GRAYSCALE);
	auto compImgDims = make_short2(sourceImage.cols, sourceImage.rows / NUM_CHANNELS);
	dim3 blockSize(compImgDims.x / THREADS_PER_BLOCK, compImgDims.y / THREADS_PER_BLOCK);
	dim3 threadSize(THREADS_PER_BLOCK, THREADS_PER_BLOCK);

	// Separate the image into the three component channels
	if (!sourceImage.isContinuous())
	{
		std::cout << "This implementation does not support discontinuous images." << std::endl;
		return DISCONT_MATRIX;
	}
	auto channelSize = compImgDims.x * compImgDims.y;
	auto blueChannel = sourceImage.ptr();
	auto greenChannel = blueChannel + channelSize;
	auto redChannel = greenChannel + channelSize;

	// Allocate space on the GPU to copy the images and other required variables to
	// For the three channel buffers, enough space is allocated to construct a full image pyramid
	// For the composite buffer, enough space is allocated to construct a 3-channel image
	Image dev_red, dev_green, dev_blue, dev_comp;
	Image dev_redEdges, dev_greenEdges, dev_blueEdges;
	short2 *dev_alignGR, *dev_alignGB;
	unsigned long long* dev_errorSum;
	CUDA_CALL(hipMalloc(&dev_red, channelSize * 4 / 3), DEV_ALLOC_FAIL, __LINE__);
	CUDA_CALL(hipMalloc(&dev_green, channelSize * 4 / 3), DEV_ALLOC_FAIL, __LINE__);
	CUDA_CALL(hipMalloc(&dev_blue, channelSize * 4 / 3), DEV_ALLOC_FAIL, __LINE__);
	CUDA_CALL(hipMalloc(&dev_redEdges, channelSize * 4 / 3), DEV_ALLOC_FAIL, __LINE__);
	CUDA_CALL(hipMalloc(&dev_greenEdges, channelSize * 4 / 3), DEV_ALLOC_FAIL, __LINE__);
	CUDA_CALL(hipMalloc(&dev_blueEdges, channelSize * 4 / 3), DEV_ALLOC_FAIL, __LINE__);
	CUDA_CALL(hipMalloc(&dev_comp, channelSize * NUM_CHANNELS), DEV_ALLOC_FAIL, __LINE__);
	CUDA_CALL(hipMalloc(&dev_alignGR, sizeof(short2)), DEV_ALLOC_FAIL, __LINE__);
	CUDA_CALL(hipMalloc(&dev_alignGB, sizeof(short2)), DEV_ALLOC_FAIL, __LINE__);
	CUDA_CALL(hipMalloc(&dev_errorSum, sizeof(unsigned long long) * NUM_ALIGN_NEIGHBORS), DEV_ALLOC_FAIL, __LINE__);

	// Copy data from the provided image into the GPU buffers
	CUDA_CALL(hipMemcpy(dev_red, redChannel, channelSize, hipMemcpyHostToDevice), DEV_CPY_FAIL, __LINE__);
	CUDA_CALL(hipMemcpy(dev_green, greenChannel, channelSize, hipMemcpyHostToDevice), DEV_CPY_FAIL, __LINE__);
	CUDA_CALL(hipMemcpy(dev_blue, blueChannel, channelSize, hipMemcpyHostToDevice), DEV_CPY_FAIL, __LINE__);
	
	// Prepare CUDA timing variables
	hipEvent_t start, edges, pyramids, alignment, finish;
	CUDA_CALL(hipEventCreate(&start), GPU_TIMING_FAIL, __LINE__);
	CUDA_CALL(hipEventCreate(&edges), GPU_TIMING_FAIL, __LINE__);
	CUDA_CALL(hipEventCreate(&pyramids), GPU_TIMING_FAIL, __LINE__);
	CUDA_CALL(hipEventCreate(&alignment), GPU_TIMING_FAIL, __LINE__);
	CUDA_CALL(hipEventCreate(&finish), GPU_TIMING_FAIL, __LINE__);

	// Detect edges and generate image pyramids
	CUDA_CALL(hipEventRecord(start, 0), GPU_TIMING_FAIL, __LINE__);
	generateImagePyramids<<<blockSize, threadSize>>>(dev_red, dev_green, dev_blue, compImgDims);
	CUDA_CALL(hipEventRecord(pyramids, 0), GPU_TIMING_FAIL, __LINE__);
	detectEdges<<<blockSize, threadSize>>>(dev_red, dev_green, dev_blue, dev_redEdges, dev_greenEdges, dev_blueEdges,
		compImgDims);
	CUDA_CALL(hipEventRecord(edges, 0), GPU_TIMING_FAIL, __LINE__);

	// Compute the alignments for the color channels
	if (multiLayerAlignMode)
	{
		// Calculate the sizes and offsets of each of the image pyramid levels
		PyramidLevel levels[NUM_ALIGN_LEVELS];
		levels[0].offset = 0;
		levels[0].dims = make_short2(compImgDims.x, compImgDims.y);
		for (short i = 1; i < NUM_ALIGN_LEVELS; ++i)
		{
			levels[i].offset = levels[i - 1].offset + (levels[i - 1].dims.x * levels[i - 1].dims.y);
			levels[i].dims = make_short2(levels[i - 1].dims.x / 2, levels[i - 1].dims.y / 2);
		}

		short2 startAlign = make_short2(0, 0);
		CUDA_CALL(hipMemcpy(dev_alignGR, &startAlign, sizeof(short2), hipMemcpyHostToDevice), DEV_CPY_FAIL, __LINE__);
		CUDA_CALL(hipMemcpy(dev_alignGB, &startAlign, sizeof(short2), hipMemcpyHostToDevice), DEV_CPY_FAIL, __LINE__);
		for (short i = NUM_ALIGN_LEVELS - 1; i >= 0; --i)
		{
			if (levels[i].dims.x < MIN_PYRAMID_SIZE || levels[i].dims.y < MIN_PYRAMID_SIZE)
				continue;

			// Calculate the margin to ignore at this pyramid level
			const unsigned int leftThreshold   = floor(levels[i].dims.x * BORDER_CUT_MARGIN);
			const unsigned int rightThreshold  = ceil(levels[i].dims.x * (1 - BORDER_CUT_MARGIN));
			const unsigned int topThreshold    = floor(levels[i].dims.y * BORDER_CUT_MARGIN);
			const unsigned int bottomThreshold = ceil(levels[i].dims.y * (1 - BORDER_CUT_MARGIN));
			short4 threshold = make_short4(leftThreshold, rightThreshold, topThreshold, bottomThreshold);

			// Perform image alignment
			dim3 pyramidBlockSize(levels[i].dims.x / THREADS_PER_BLOCK, levels[i].dims.y / THREADS_PER_BLOCK);
			alignImages<<<pyramidBlockSize, threadSize>>>(dev_greenEdges + levels[i].offset,
				dev_redEdges + levels[i].offset, levels[i].dims, dev_alignGR, threshold, dev_errorSum);
			alignImages<<<pyramidBlockSize, threadSize>>>(dev_greenEdges + levels[i].offset,
				dev_blueEdges + levels[i].offset, levels[i].dims, dev_alignGB, threshold, dev_errorSum);
		}
	}
	else
	{
		short2 bestAlignmentGR = make_short2(0, 0);
		short2 bestAlignmentGB = make_short2(0, 0);
		unsigned long long error[2];
		unsigned long long bestErrorGR = error[0];
		unsigned long long bestErrorGB = error[1];

		// Calculate the margin to ignore
		const unsigned int leftThreshold   = floor(compImgDims.x * BORDER_CUT_MARGIN);
		const unsigned int rightThreshold  = ceil(compImgDims.x * (1 - BORDER_CUT_MARGIN));
		const unsigned int topThreshold    = floor(compImgDims.y * BORDER_CUT_MARGIN);
		const unsigned int bottomThreshold = ceil(compImgDims.y * (1 - BORDER_CUT_MARGIN));
		short4 threshold = make_short4(leftThreshold, rightThreshold, topThreshold, bottomThreshold);

		short2 trialAlignment;
		for (short i = -alignmentWindow.x / 2; i < alignmentWindow.x / 2; ++i)
		{
			for (short j = -alignmentWindow.y / 2; j < alignmentWindow.y / 2; ++j)
			{
				trialAlignment = make_short2(i, j);
				scoreAlignment<<<blockSize, threadSize>>>(dev_greenEdges, dev_redEdges, compImgDims, trialAlignment,
					threshold, dev_errorSum);
				scoreAlignment<<<blockSize, threadSize>>>(dev_greenEdges, dev_blueEdges, compImgDims, trialAlignment,
					threshold, dev_errorSum + 1);

				CUDA_CALL(hipMemcpy(&error, dev_errorSum, sizeof(unsigned long long) * 2, hipMemcpyDeviceToHost),
					HOST_CPY_FAIL, __LINE__);

				if (error[0] < bestErrorGR)
				{
					bestErrorGR = error[0];
					bestAlignmentGR = make_short2(i, j);
				}
				if (error[1] < bestErrorGB)
				{
					bestErrorGB = error[1];
					bestAlignmentGB = make_short2(i, j);
				}
			}
		}

		// Copy the computed alignment to the device
		CUDA_CALL(hipMemcpy(dev_alignGR, &bestAlignmentGR, sizeof(short2), hipMemcpyHostToDevice), DEV_CPY_FAIL,
			__LINE__);
		CUDA_CALL(hipMemcpy(dev_alignGB, &bestAlignmentGB, sizeof(short2), hipMemcpyHostToDevice), DEV_CPY_FAIL,
			__LINE__);
	}

	// Finish producing the composite and perform post-processing
	CUDA_CALL(hipEventRecord(alignment, 0), GPU_TIMING_FAIL, __LINE__);
	produceComposite<<<blockSize, threadSize>>>(dev_red, dev_green, dev_blue, dev_comp, dev_alignGR, dev_alignGB,
		compImgDims);
	CUDA_CALL(hipEventRecord(finish, 0), GPU_TIMING_FAIL, __LINE__);
	CUDA_CALL(hipEventSynchronize(finish), GPU_TIMING_FAIL, __LINE__);

	// Record the elapsed timing
	float pyramidTime, edgeTime, alignmentTime, compositingTime, totalTime;
	CUDA_CALL(hipEventElapsedTime(&pyramidTime, start, pyramids), GPU_TIMING_FAIL, __LINE__);
	CUDA_CALL(hipEventElapsedTime(&edgeTime, pyramids, edges), GPU_TIMING_FAIL, __LINE__);
	CUDA_CALL(hipEventElapsedTime(&alignmentTime, edges, alignment), GPU_TIMING_FAIL, __LINE__);
	CUDA_CALL(hipEventElapsedTime(&compositingTime, alignment, finish), GPU_TIMING_FAIL, __LINE__);
	CUDA_CALL(hipEventElapsedTime(&totalTime, start, finish), GPU_TIMING_FAIL, __LINE__);

	// Copy the composite back from the device
	auto compBuf = malloc(channelSize * NUM_CHANNELS);
	CUDA_CALL(hipMemcpy(compBuf, dev_comp, channelSize * NUM_CHANNELS, hipMemcpyDeviceToHost), HOST_CPY_FAIL, __LINE__);
	cv::Mat compositeImage(compImgDims.y, compImgDims.x, CV_8UC3, compBuf);

	// Print timings and alignments
	short2 grAlign, gbAlign;
	CUDA_CALL(hipMemcpy(&grAlign, dev_alignGR, sizeof(short2), hipMemcpyDeviceToHost), HOST_CPY_FAIL, __LINE__);
	CUDA_CALL(hipMemcpy(&gbAlign, dev_alignGB, sizeof(short2), hipMemcpyDeviceToHost), HOST_CPY_FAIL, __LINE__);
	auto endTime = std::chrono::high_resolution_clock::now();
	auto elapsedTime = std::chrono::duration_cast<std::chrono::duration<float>>(endTime - startTime);
	std::cout << "Alignment computed:" << std::endl;
	std::cout << "    Green/Red:  (" << grAlign.x << ", " << grAlign.y << ")" << std::endl;
	std::cout << "    Green/Blue: (" << gbAlign.x << ", " << gbAlign.y << ")" << std::endl;
	std::cout << "Composition took " << elapsedTime.count() << " seconds." << std::endl;
	std::cout << "    Pyramid creation time: " << pyramidTime << " ms" << std::endl;
	std::cout << "    Edge detection time: " << edgeTime << " ms" << std::endl;
	std::cout << "    Alignment time: " << alignmentTime << " ms" << std::endl;
	std::cout << "    Compositing time: " << compositingTime << " ms" << std::endl;
	std::cout << std::endl;
	std::cout << "  Total GPU time: " << totalTime << " ms" << std::endl;

	// Free CUDA timing variables
	CUDA_CALL(hipEventDestroy(start), GPU_TIMING_FAIL, __LINE__);
	CUDA_CALL(hipEventDestroy(edges), GPU_TIMING_FAIL, __LINE__);
	CUDA_CALL(hipEventDestroy(pyramids), GPU_TIMING_FAIL, __LINE__);
	CUDA_CALL(hipEventDestroy(alignment), GPU_TIMING_FAIL, __LINE__);
	CUDA_CALL(hipEventDestroy(finish), GPU_TIMING_FAIL, __LINE__);

	// Free all allocated device memory
	hipFree(dev_red);
	hipFree(dev_green);
	hipFree(dev_blue);
	hipFree(dev_redEdges);
	hipFree(dev_greenEdges);
	hipFree(dev_blueEdges);
	hipFree(dev_comp);
	hipFree(dev_alignGR);
	hipFree(dev_alignGB);
	hipFree(dev_errorSum);

	// Generate a reduced-size version of the image to display more easily
	cv::Mat compSmall;
	auto smallSize = compImgDims.x > compImgDims.y
		? cv::Size(MAX_SMALL_IMG_DIM, (short)(compImgDims.y * (((float) MAX_SMALL_IMG_DIM) / compImgDims.x)))
		: cv::Size((short)(compImgDims.x * (((float) MAX_SMALL_IMG_DIM) / compImgDims.y)), MAX_SMALL_IMG_DIM);
	cv::resize(compositeImage, compSmall, smallSize);

	// Save a full-size and small version of the composite image
	cv::imwrite(std::string(imageName) + ".bmp", compositeImage);
	cv::imwrite(std::string(imageName) + "_small.bmp", compSmall);

	// Display the resulting image
	cv::imshow(imageName, compSmall);
	cv::waitKey();

	// Free remaining memory
	free(compBuf);

	return SUCCESS;
}